#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iostream>
#include <string>
#include <time.h>

/**
* Rokas Palionis IFF-5/8 individualus darbas
* 


#define min(a, b) (a < b ? a : b)

/**
* Sujungia duomenis, kiekvienai duomen� daliai
*/
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
	long i = start;
	long j = middle;
	for (long k = start; k < end; k++) {
		if (i < middle && (j >= end || source[i] < source[j])) {
			dest[k] = source[i];
			i++;
		}
		else {
			dest[k] = source[j];
			j++;
		}
	}
}
/**
* Suskai�iuoja dabartin�s gijos id
*/
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
	int x;
	return threadIdx.x +
		threadIdx.y * (x = threads->x) +
		threadIdx.z * (x *= threads->y) +
		blockIdx.x  * (x *= threads->z) +
		blockIdx.y  * (x *= blocks->z) +
		blockIdx.z  * (x *= blocks->y);
}
/**
* Pateiktai duomen� daliai vykdo MergeSort rikiavim�
*/
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
	unsigned int idx = getIdx(threads, blocks);
	long start = width*idx*slices,
		middle,
		end;

	for (long slice = 0; slice < slices; slice++) {
		if (start >= size)
			break;

		middle = min(start + (width >> 1), size);
		end = min(start + width, size);
		gpu_bottomUpMerge(source, dest, start, middle, end);
		start += width;
	}
}
/**
* Pradeda vykdyti MergeSort algoritm�: paruo�ia atmint�, perkopijuoja duomenis � GPU
* paleid�ia gpu_mergesort rikiavim� kiekvienai duomen� porcijai
*/
void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

	// Sukuriami du masyvai
	// Vykdymo metu jie yra kei�iami vienas su kitu
	long* D_data;
	long* D_swp;
	// kiekis
	dim3* D_threads;
	dim3* D_blocks;

	// paskiriama atmintis masyvams i GPU
	hipMalloc((void**)&D_data, size * sizeof(long));
	hipMalloc((void**)&D_swp, size * sizeof(long));

	hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&D_threads, sizeof(dim3));
	hipMalloc((void**)&D_blocks, sizeof(dim3));

	// is gpu i cpu
	hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
	hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

	long* A = D_data;
	long* B = D_swp;

	long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
		blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

	clock_t startTime, endTime;

	// Vykdomas MergeSort algoritmas
	startTime = clock();

	for (int width = 2; width < (size << 1); width <<= 1) {
		long slices = size / ((nThreads)* width) + 1;

		gpu_mergesort << <blocksPerGrid, threadsPerBlock >> >(A, B, size, width, slices, D_threads, D_blocks);

		// Switch the input / output arrays instead of copying them around
		A = A == D_data ? D_swp : D_data;
		B = B == D_data ? D_swp : D_data;
	}

	hipDeviceSynchronize();

	endTime = clock();
	std::cout << "\n\tVykdymo laikas (merge sort): " << endTime - startTime << " ms\n";

	// grazinam i cpu
	hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

	hipFree(A);
	hipFree(B);
}

void generateData(long array[], long length) {
	
	for (long i = 0; i < length; i++)
		array[i] = rand();
}

void readData(long data[], long size, std::ifstream & readStream) {

	for (long i = 0; readStream.good() && i < size; i++) {
		readStream >> data[i];
	}

	return;
}

void writeData(long data[], long size, std::ofstream & writeStream) {

	for (long i = 0; writeStream.good() && i < size; i++) {
		writeStream << data[i] << std::endl;
	}

	return;
}



int main() {



	dim3 threadsPerBlock;
	dim3 blocksPerGrid;

	threadsPerBlock.x = 1;
	threadsPerBlock.y = 1;
	threadsPerBlock.z = 1;

	blocksPerGrid.x = 1;
	blocksPerGrid.y = 1;
	blocksPerGrid.z = 1;

	long size = 600000;

	long *data = new long[size];

	std::cout << "�veskite duomen� kiek� (n > 0), gij� kiek� (k >= 1), blok� kiek� (l >= 1): ";
	std::cin >> size >> threadsPerBlock.x >> blocksPerGrid.x;

	if (size < 1) {
		std::cout << "\n\tElement� kiekis turi b�ti didesnis u� 0!\n\nPrograma nes�kmingai baig� darb�!\n";
		return -1;
	}

	std::string readFileName = "data/" + std::to_string(size) + ".data.txt";
	std::string writeFileName = "results/" + std::to_string(size) + ".result.txt";

	std::ifstream readStream(readFileName);

	// Tikrinama ar failas egzistuoja
	if (readStream.good()) {
		readData(data, size, readStream);
		readStream.close();
	}
	else {
		generateData(data, size);

		// generuota masyva spausdina i faila
		std::ofstream writeStreamData(readFileName);
		writeData(data, size, writeStreamData);
		writeStreamData.close();
	}

	mergesort(data, size, threadsPerBlock, blocksPerGrid);

	std::ofstream writeStream(writeFileName);

	writeData(data, size, writeStream);

	writeStream.close();

	std::cout << "\nPrograma s�kmingai baig� darb�!\n";

	return 0;
}