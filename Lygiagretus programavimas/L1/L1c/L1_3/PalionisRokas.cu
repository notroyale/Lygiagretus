/*
* Rokas Palionis IFF-5/8
*
* 1. Kokia tvarka startuoja procesai? Galimi atsakymo variantai: tokia, kokia u�ra�yti, atsitiktine, atvirk��ia.
* 		Atsakymas: atsitiktine.
* 2. Kokia tvarka vykdomi procesai? Galimi atsakymo variantai: tokia, kokia startuoja, atsitiktine, atvirk��ia.
* 		Atsakymas: atsitiktine.
* 3. Kiek iteracij� i� eil�s padaro vienas procesas? Galimi atsakymo variantai: vienos dal�, vien� pilnai, visas,
* atsitiktin� skai�i�.
* 		Atsakymas: visas.
* 4. Kokia tvarka to paties duomen� masyvo elementai sura�omi � rezultat� masyv�? Galimi atsakymo variantai:
* tokia, kokia sura�yti duomen� masyve, atsitiktine, atvirk��ia.
* 		Atsakymas: atsitiktine.
* 5. Kurioje programoje trumpiausias vienos gijos kodas?
*		Atsakymas: C++ & OpenMP
* 6. Kokiu kompiuteriu vykd�te savo programas? Nurodykite branduoli� skai�i� ir da�nius, OA apimt�, OS, NVIDIA plok�t�s tip�.
*		Atsakymas: i5-7300HQ 4 (8 HT) 2.6Ghz (3.6Ghz Turbo Boost), 8GB DDR3, Windows 10 Education 64-Bit, GTX 1050 2GB GDDR5.
*/

#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include "hip/hip_runtime.h"


using namespace std;

//Masyvo element� dydis
const unsigned int array_size = 20;
//Gij� skai�ius
int thread_amount = 0;

//Bendra strukt�ra P
struct ThreadStruct {
	int thread_number;
	char name[20];
	int wins;
	double points;
};

/*
* Nuskaito duomenis i� duomen� failo � masyvus.
*
* @param S - char tipo masyvas
* @param I - int tipo masyvas
* @param D - double tipo masyvas
*/
void ReadData(char S[array_size][array_size], int I[], double D[]);
/*
* I�veda duomenis i� masyv� � rezultat� fail�.
*
* @param S - char tipo masyvas
* @param I - int tipo masyvas
* @param D - double tipo masyvas
*/
void WriteData(char S[array_size][array_size], int I[], double D[]);
/*
* I�veda duomenis i� bendro gij� masyvo � rezultat� fail�.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @return - void
*/
void WriteThreads(ThreadStruct P[], int thread_identifier);
/*
* Vykdomas element� paruo�imas darbui su CUDA. Taip pat �ia paleid�iama pati CUDA.
*
* @param S - char tipo masyvas
* @param I - int tipo masyvas
* @param D - double tipo masyvas
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @return - cudaError_t
*/
hipError_t writeWithCuda(char S[array_size][array_size], int I[], double D[], ThreadStruct P[], int *thread_identifier);
/*
* Vykdo CUDA funkcij�, kuri kiekvienai skirtingai gijai paskiria skirting� veiksm�.
*
* @param S - char tipo masyvas
* @param I - int tipo masyvas
* @param D - double tipo masyvas
* @param P - ThreadStruct tipo masyvas
* @param dev_id - P masyvo element� identifikatorius
* @return - void
*/
__global__ void writeKernel(char S[array_size * array_size], int I[], double D[], ThreadStruct P[], int *dev_id);
/*
* Kiekviena gija �ra�o savo elementus � bendr� masyv� P.
*
* @param S - char tipo masyvas
* @param I - int tipo masyvas
* @param D - double tipo masyvas
* @param P - ThreadStruct tipo masyvas
* @param dev_id - P masyvo element� identifikatorius
* @param thread_number - vykdomos gijos identifikatorius
* @return - void
*/
__device__ void writeDevice(char S[array_size * array_size], int I[], double D[], ThreadStruct P[], int element_id, int thread_number);

int main() {

	//Nustato lokalizacij�, kad tinkamai b�t� nuskaitomi lietuvi�ki ra�menys
	setlocale(LC_ALL, "");

	//Duomen� masyvai: S - string, I - int, D - double
	char S[array_size][array_size];
	int I[array_size];
	double D[array_size];

	//Bendras masyvas sudarytas i� strukt�ros ThreadStruct ir reik�m�m sud�ti papildomas masyvo identifikatorius
	ThreadStruct P[array_size];
	int thread_identifier = 0;

	//Vykdomos funkcijos: nuskaityti duomenis � masyvus, tada tuos duomenis i�vesti � duomen� fail�
	ReadData(S, I, D);
	WriteData(S, I, D);

	//Paleid�iama CUDA
	hipError_t cudaStatus = writeWithCuda(S, I, D, P, &thread_identifier);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "writeWithCuda failed!");
		return 1;
	}
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//�ra�omi bendro masyvo duomenys (po CUDA vykdymo) � rezultat� fail�
	//WriteThreads(P, thread_identifier);
	WriteThreads(P, thread_identifier);

	cout << "Programa baig� darb�!" << endl;

    return 0;
}



void ReadData(char S[array_size][array_size], int I[], double D[]) {

	ifstream  readStream("PalionisRokas_L1c.dat.txt");

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (20), juos �ra�o � masyv�
	for (int i = 0; !readStream.eof() && i < array_size; i++) {
		readStream >> S[i] >> I[i] >> D[i];
		thread_amount++;
	}

	readStream.close();

	return;
}

void WriteData(char S[array_size][array_size], int I[], double D[]) {

	ofstream writeStream("PalionisRokas_L1c.rez.txt");

	//I�vedamos stulpeli� antra�t�s
	writeStream << setw(3) << "Nr." << " " << setw(14) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (20), juos �ra�o � fail� pagal stulpelius
	for (int i = 0; S[i] != "" && I[i] >= 0 && D[i] >= 0 && i < array_size; i++) {
		writeStream << setw(2) << right << i + 1 << ") " << setw(array_size) << left << S[i] << " " << setw(2) << right << I[i] << "  " << setw(5) << right << fixed << setprecision(2) << D[i] << endl;
	}

	//I�vedama papildoma tu��ia eilut�
	writeStream << endl;

	writeStream.close();

	return;
}

void WriteThreads(ThreadStruct P[], int thread_identifier) {

	ofstream writeStream("PalionisRokas_L1c.rez.txt", fstream::app);

	//I�vedamos stulpeli� antra�t�s
	writeStream << setw(3) << "Nr." << " " << setw(4) << "Gija" << " " << setw(14) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkam� reik�mi� P masyve, juos �ra�o � fail� pagal stulpelius
	for (int i = 0; P[i].thread_number >= 0 && P[i].name != "" && P[i].wins >= 0 && P[i].points >= 0 && i < thread_identifier; i++) {
		writeStream << setw(2) << right << i + 1 << ")   " << setw(2) << right << P[i].thread_number << " " << setw(array_size) << left << P[i].name << " " << setw(2) << right << P[i].wins << "  " << setw(5) << right << fixed << setprecision(2) << P[i].points << endl;
	}

	writeStream.close();

	return;
}

hipError_t writeWithCuda(char S[array_size][array_size], int I[], double D[], ThreadStruct P[], int *thread_identifier) {

	char *dev_a;
	int *dev_b = 0;
	double *dev_c = 0;
	ThreadStruct *dev_d;

	int *dev_id = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for four arrays (three input, one output).
	cudaStatus = hipMalloc((void**)&dev_d, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, array_size * array_size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, array_size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, array_size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for one variable (array P identifier).
	cudaStatus = hipMalloc((void**)&dev_id, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, S, array_size * array_size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, I, array_size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, D, array_size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	writeKernel<<<1, thread_amount>>>(dev_a, dev_b, dev_c, dev_d, dev_id);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "writeernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(P, dev_d, array_size * sizeof(ThreadStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(thread_identifier, dev_id, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_d);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_id);

	return cudaStatus;
}

__global__ void writeKernel(char S[array_size * array_size], int I[], double D[], ThreadStruct P[], int *dev_id) {

	//CUDA gijos numeris
	int thread_number = threadIdx.x;
	//Elemento masyve P identifikatorius
	int element_id = atomicAdd(dev_id, 1);
	//Kiekviena gija �ra�in�ja savo duomenis
	writeDevice(S, I, D, P, element_id, thread_number);

	return;
}

__device__ void writeDevice(char S[array_size * array_size], int I[], double D[], ThreadStruct P[], int element_id, int thread_number) {

	//�ra�omi duomenys � bendr� masyv�
	P[element_id].thread_number = thread_number;
	int start_identifier = thread_number * array_size;
	for (int i = 0; i < array_size; i++) {
		P[element_id].name[i] = S[start_identifier++];
	}
	P[element_id].wins = I[thread_number];
	P[element_id].points = D[thread_number];

	return;
}