/*
* Mantvydas Zakarevi�ius IFF-4/2 L4a
*
*/

#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include "hip/hip_runtime.h"


using namespace std;

//Masyvo element� dydis
const unsigned int array_size = 30;
//Gij� skai�ius
int thread_amount = 12;

//Skaitymo failas
string readFileName = "ZakareviciusM_L4a.dat.txt";
//Spausdinimo failas
string writeFileName = "ZakareviciusM_L4a.rez.txt";

//Bendra strukt�ra P
struct ThreadStruct {
	//int thread_number;
	char name[20 * 5];
	int wins;
	double points;
};

/*
* Nuskaito duomenis i� duomen� failo � masyvus.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param readStream - skaitymo srautas
* @return - void
*/
void ReadData(ThreadStruct P[array_size], int & p_amount, ifstream & readStream);
/*
* I�veda duomenis i� masyv� � rezultat� fail�.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param writeStream - spausdinimo srautas
* @return - void
*/
void WriteData(ThreadStruct P[array_size], int p_amount, ofstream & writeStream, int element);
/*
* I�veda duomenis i� bendro gij� masyvo � rezultat� fail�.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param writeStream - spausdinimo srautas
* @return - void
*/
void WriteThreads(ThreadStruct P[], int p_amount, ofstream & writeStream);
/*
* Vykdomas element� paruo�imas darbui su CUDA. Taip pat �ia paleid�iama pati CUDA.
*
* @param P1 - ThreadStruct tipo masyvas
* @param p1_amount - P1 masyvo element� identifikatorius
* @param P2 - ThreadStruct tipo masyvas
* @param p2_amount - P2 masyvo element� identifikatorius
* @param P3 - ThreadStruct tipo masyvas
* @param p3_amount - P3 masyvo element� identifikatorius
* @param P4 - ThreadStruct tipo masyvas
* @param p4_amount - P4 masyvo element� identifikatorius
* @param P5 - ThreadStruct tipo masyvas
* @param p5_amount - P5 masyvo element� identifikatorius
* @param P - ThreadStruct tipo masyvas
* @param p_amount - P masyvo element� identifikatorius
* @return - cudaError_t
*/
hipError_t writeWithCuda(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount);
/*
* Vykdo CUDA funkcij�, kuri kiekvienai skirtingai gijai paskiria skirting� veiksm�.
*
* @param P1 - ThreadStruct tipo masyvas
* @param p1_amount - P1 masyvo element� identifikatorius
* @param P2 - ThreadStruct tipo masyvas
* @param p2_amount - P2 masyvo element� identifikatorius
* @param P3 - ThreadStruct tipo masyvas
* @param p3_amount - P3 masyvo element� identifikatorius
* @param P4 - ThreadStruct tipo masyvas
* @param p4_amount - P4 masyvo element� identifikatorius
* @param P5 - ThreadStruct tipo masyvas
* @param p5_amount - P5 masyvo element� identifikatorius
* @param P - ThreadStruct tipo masyvas
* @param p_amount - P masyvo element� identifikatorius
* @return - void
*/
__global__ void writeKernel(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount);
/*
* Kiekviena gija �ra�o savo elementus � bendr� masyv� P.
*
* @param P1 - ThreadStruct tipo masyvas
* @param p1_amount - P1 masyvo element� identifikatorius
* @param P2 - ThreadStruct tipo masyvas
* @param p2_amount - P2 masyvo element� identifikatorius
* @param P3 - ThreadStruct tipo masyvas
* @param p3_amount - P3 masyvo element� identifikatorius
* @param P4 - ThreadStruct tipo masyvas
* @param p4_amount - P4 masyvo element� identifikatorius
* @param P5 - ThreadStruct tipo masyvas
* @param p5_amount - P5 masyvo element� identifikatorius
* @param P - ThreadStruct tipo masyvas
* @param p_amount - P masyvo element� identifikatorius
* @param thread_number - vykdomos gijos identifikatorius
* @return - void
*/
__device__ void writeDevice(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount, int thread_number);
/*
* strcpy funkcija veikianti ant CUDA gij� (C kalbos).
*
* @param dest - � kur� char masyv� kopijuojama 
* @param src - i� kurio char masyvo kopijuojama 
* @return - char*
*/
__device__ char * my_strcpy(char *dest, const char *src);
/*
* strcat funkcija veikianti ant CUDA gij� (C kalbos).
*
* @param dest - � kur� char masyv� kopijuojama
* @param src - i� kurio char masyvo kopijuojama
* @return - char*
*/
__device__ char * my_strcat(char *dest, const char *src);

int main() {

	//Nustato lokalizacij�, kad tinkamai b�t� nuskaitomi lietuvi�ki ra�menys
	setlocale(LC_ALL, "");

	//Sukuriami masyvai
	ThreadStruct P1[array_size];
	ThreadStruct P2[array_size];
	ThreadStruct P3[array_size];
	ThreadStruct P4[array_size];
	ThreadStruct P5[array_size];
	//Sukuriami masyv� kiekio identifikatoriai
	int p1_amount = 0;
	int p2_amount = 0;
	int p3_amount = 0;
	int p4_amount = 0;
	int p5_amount = 0;

	//Bendras masyvas sudarytas i� strukt�ros ThreadStruct ir reik�m�m sud�ti papildomas masyvo identifikatorius
	ThreadStruct P[array_size];
	int p_amount = 0;

	//Skaitomi duomenys � masyvus
	ifstream readStream(readFileName);
	ReadData(P1, p1_amount, readStream);
	ReadData(P2, p2_amount, readStream);
	ReadData(P3, p3_amount, readStream);
	ReadData(P4, p4_amount, readStream);
	ReadData(P5, p5_amount, readStream);
	readStream.close();

	//Paleid�iama CUDA (vykdomos gijos)
	hipError_t cudaStatus = writeWithCuda(P1, p1_amount, P2, p2_amount, P3, p3_amount, P4, p4_amount, P5, p5_amount, P, &p_amount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "writeWithCuda failed!");
		return 1;
	}
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//Spausdinami pradiniai duomenys
	ofstream writeStream(writeFileName);
	WriteData(P1, p1_amount, writeStream, 1);
	WriteData(P2, p2_amount, writeStream, 2);
	WriteData(P3, p3_amount, writeStream, 3);
	WriteData(P4, p4_amount, writeStream, 4);
	WriteData(P5, p5_amount, writeStream, 5);
	writeStream.close();

	//Spausdinamas bendras masyvas
	writeStream.open(writeFileName, fstream::app);
	WriteThreads(P, p_amount, writeStream);
	writeStream.close();

	cout << "Programa baig� darb�!" << endl;

    return 0;
}



void ReadData(ThreadStruct P[array_size], int & p_amount, ifstream & readStream) {

	//Element� kiekvienoje sekcijoje skai�ius
	int elements_amount;

	readStream >> skipws >> elements_amount;
	if (elements_amount > 0) {
		//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_element_limit), juos �ra�o � masyv�
		for (int i = 0; readStream.good() && i < array_size && i < elements_amount; i++) {
			readStream >> P[i].name >> P[i].wins >> P[i].points;
			p_amount++;
		}
	}

	return;
}

void WriteData(ThreadStruct P[array_size], int p_amount, ofstream & writeStream, int element) {

	//I�vedamos stulpeli� antra�t�s
	writeStream << element << " ---------------------------------\n";
	writeStream << setw(3) << "Nr." << " " << setw(14) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_limit), juos �ra�o � fail� pagal stulpelius
	for (int i = 0; P[i].name != "" && P[i].wins >= 0 && P[i].points >= 0 && i < p_amount; i++) {
		writeStream << setw(2) << right << i + 1 << ") " << setw(20) << left << P[i].name << " " << setw(2) << right << P[i].wins << "  " << setw(5) << right << fixed << setprecision(2) << P[i].points << endl;
	}

	//I�vedama papildoma tu��ia eilut�
	writeStream << endl;

	return;
}

void WriteThreads(ThreadStruct P[], int p_amount, ofstream & writeStream) {

	//I�vedamos stulpeli� antra�t�s
	writeStream << " ----------------------------------\n";
	writeStream << setw(3) << "Nr." << " " << setw(60) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_limit), juos �ra�o � fail� pagal stulpelius
	for (int i = 0; P[i].name != "" && P[i].wins >= 0 && P[i].points >= 0 && i < p_amount; i++) {
		writeStream << setw(2) << right << i + 1 << ") " << setw(60+6) << left << P[i].name << " " << setw(2) << right << P[i].wins << "  " << setw(5) << right << fixed << setprecision(2) << P[i].points << endl;
	}

	//I�vedama papildoma tu��ia eilut�
	writeStream << endl;

	return;
}

hipError_t writeWithCuda(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount) {

	ThreadStruct *dev_P1;
	ThreadStruct *dev_P2;
	ThreadStruct *dev_P3;
	ThreadStruct *dev_P4;
	ThreadStruct *dev_P5;
	ThreadStruct *dev_P;

	int *dev_p1_amount = 0;
	int *dev_p2_amount = 0;
	int *dev_p3_amount = 0;
	int *dev_p4_amount = 0;
	int *dev_p5_amount = 0;
	int *dev_p_amount = 0;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for four arrays (three input, one output).
	cudaStatus = hipMalloc((void**)&dev_P1, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P2, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P3, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P4, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P5, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P, array_size * sizeof(ThreadStruct));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for arrays identifiers
	cudaStatus = hipMalloc((void**)&dev_p1_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p2_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p3_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p4_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p5_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p_amount, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_P1, P1, array_size * sizeof(ThreadStruct), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P2, P2, array_size * sizeof(ThreadStruct), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P3, P3, array_size * sizeof(ThreadStruct), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P4, P4, array_size * sizeof(ThreadStruct), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P5, P5, array_size * sizeof(ThreadStruct), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Assigns arrays amounts to CUDA memory variables
	*dev_p1_amount = p1_amount;
	*dev_p2_amount = p2_amount;
	*dev_p3_amount = p3_amount;
	*dev_p4_amount = p4_amount;
	*dev_p5_amount = p5_amount;

	// Launch a kernel on the GPU with one thread for each element.
	writeKernel<<<1, thread_amount>>>(dev_P1, *dev_p1_amount, dev_P2, *dev_p2_amount, dev_P3, *dev_p3_amount, dev_P4, *dev_p4_amount, dev_P5, *dev_p5_amount, dev_P, dev_p_amount);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "writeernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(P, dev_P, array_size * sizeof(ThreadStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(p_amount, dev_p_amount, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_P);
	hipFree(dev_P1);
	hipFree(dev_P2);
	hipFree(dev_P3);
	hipFree(dev_P4);
	hipFree(dev_P5);
	hipFree(dev_p_amount);
	hipFree(dev_p1_amount);
	hipFree(dev_p2_amount);
	hipFree(dev_p3_amount);
	hipFree(dev_p4_amount);
	hipFree(dev_p5_amount);

	return cudaStatus;
}

__global__ void writeKernel(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount) {

	//CUDA gijos numeris
	int thread_number = threadIdx.x;

	//Padidina element� kiek�
	atomicAdd(p_amount, 1);

	//Kiekviena gija �ra�in�ja savo duomenis
	writeDevice(P1, p1_amount, P2, p2_amount, P3, p3_amount, P4, p4_amount, P5, p5_amount, P, p_amount, thread_number);

	return;
}

__device__ void writeDevice(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int *p_amount, int thread_number) {

	//printf("Thread id: %d!\n", thread_number);

	//�ra�omi duomenys � bendr� masyv�
	my_strcat(P[thread_number].name, P1[thread_number].name);
	my_strcat(P[thread_number].name, P2[thread_number].name);
	my_strcat(P[thread_number].name, P3[thread_number].name);
	my_strcat(P[thread_number].name, P4[thread_number].name);
	my_strcat(P[thread_number].name, P5[thread_number].name);
	P[thread_number].wins += P1[thread_number].wins + P2[thread_number].wins + P3[thread_number].wins + P4[thread_number].wins + P5[thread_number].wins;
	P[thread_number].points += P1[thread_number].points + P2[thread_number].points + P3[thread_number].points + P4[thread_number].points + P5[thread_number].points;

	return;
}

__device__ char * my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

__device__ char * my_strcat(char *dest, const char *src) {
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest + i, src);
	return dest;
}