/*
* Rokas Palionis IFF-5/8 L4b
*
*/

#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include "hip/hip_runtime.h"
#include ""
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

using namespace std;

//Masyvo element� dydis
const unsigned int array_size = 30;

//Skaitymo failas
string readFileName = "PalionisR_L4b.dat.txt";
//Spausdinimo failas
string writeFileName = "PalionisR_L4b.rez.txt";

//Bendra strukt�ra P
struct ThreadStruct {
	char name[20 * 5] = { 0 };
	int wins;
	double points;
};

/*
* Nuskaito duomenis i� duomen� failo � masyvus.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param readStream - skaitymo srautas
* @param elements_amount - element� kiekvienoje grup�je skai�ius
* @return - void
*/
void ReadData(ThreadStruct P[array_size], int & p_amount, ifstream & readStream, int elements_amount);
/*
* I�veda duomenis i� masyv� � rezultat� fail�.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param writeStream - spausdinimo srautas
* @return - void
*/
void WriteData(ThreadStruct P[array_size], int p_amount, ofstream & writeStream, int element);
/*
* I�veda duomenis i� bendro gij� masyvo � rezultat� fail�.
*
* @param P - ThreadStruct tipo masyvas
* @param thread_identifier - P masyvo element� identifikatorius
* @param writeStream - spausdinimo srautas
* @return - void
*/
void WriteThreads(ThreadStruct P[], int p_amount, ofstream & writeStream);

void CopyArrayToVector(thrust::host_vector<ThreadStruct> CPU_P, ThreadStruct P[]);
void CopyVectorToArray(ThreadStruct P[], thrust::host_vector<ThreadStruct> CPU_P, int p_amount);

void ExecuteThrust(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int p_amount);

void CopyArrayToTypeString(string P_STRING[], ThreadStruct P[]);
void CopyArrayToTypeInt(int P_INT[], ThreadStruct P[]);
void CopyArrayToTypeDouble(double P_DOUBLE[], ThreadStruct P[]);

void ExecuteStringAdd(string P_STRING[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]);
void ExecuteIntAdd(int P_INT[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]);
void ExecuteDoubleAdd(double P_DOUBLE[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]);

void CopyTypeDataToArray(string P_STRING[], int P_INT[], double P_DOUBLE[], ThreadStruct P[]);

int main() {

	//Nustato lokalizacij�, kad tinkamai b�t� nuskaitomi lietuvi�ki ra�menys
	setlocale(LC_ALL, "");

	//Sukuriami masyvai
	ThreadStruct P1[array_size];
	ThreadStruct P2[array_size];
	ThreadStruct P3[array_size];
	ThreadStruct P4[array_size];
	ThreadStruct P5[array_size];
	//Sukuriami masyv� kiekio identifikatoriai
	int p1_amount = 0;
	int p2_amount = 0;
	int p3_amount = 0;
	int p4_amount = 0;
	int p5_amount = 0;

	//Bendras masyvas sudarytas i� strukt�ros ThreadStruct ir reik�m�m sud�ti papildomas masyvo identifikatorius
	ThreadStruct P[array_size];
	int p_amount = 0;

	//Skaitomi duomenys � masyvus
	ifstream readStream(readFileName);
	//Element� kiekvienoje sekcijoje skai�ius
	int elements_amount;
	readStream >> skipws >> elements_amount;
	ReadData(P1, p1_amount, readStream, elements_amount);
	ReadData(P2, p2_amount, readStream, elements_amount);
	ReadData(P3, p3_amount, readStream, elements_amount);
	ReadData(P4, p4_amount, readStream, elements_amount);
	ReadData(P5, p5_amount, readStream, elements_amount);
	readStream.close();

	//P bendro masyvo element� kiekis lygus kiekvienos grup�s element� skai�iui
	p_amount = elements_amount;

	//Vykdomos thrust funkcijos
	ExecuteThrust(P1, p1_amount, P2, p2_amount, P3, p3_amount, P4, p4_amount, P5, p5_amount, P, p_amount);

	//Spausdinami pradiniai duomenys
	ofstream writeStream(writeFileName);
	WriteData(P1, p1_amount, writeStream, 1);
	WriteData(P2, p2_amount, writeStream, 2);
	WriteData(P3, p3_amount, writeStream, 3);
	WriteData(P4, p4_amount, writeStream, 4);
	WriteData(P5, p5_amount, writeStream, 5);
	writeStream.close();

	//Spausdinamas bendras masyvas
	writeStream.open(writeFileName, fstream::app);
	WriteThreads(P, p_amount, writeStream);
	writeStream.close();

	cout << "Programa baig� darb�!" << endl;

    return 0;
}



void ReadData(ThreadStruct P[array_size], int & p_amount, ifstream & readStream, int elements_amount) {

	if (elements_amount > 0) {
		//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_element_limit), juos �ra�o � masyv�
		for (int i = 0; readStream.good() && i < array_size && i < elements_amount; i++) {
			readStream >> P[i].name >> P[i].wins >> P[i].points;
			p_amount++;
		}
	}

	return;
}

void WriteData(ThreadStruct P[array_size], int p_amount, ofstream & writeStream, int element) {

	//I�vedamos stulpeli� antra�t�s
	writeStream << element << " ---------------------------------\n";
	writeStream << setw(3) << "Nr." << " " << setw(14) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_limit), juos �ra�o � fail� pagal stulpelius
	for (int i = 0; P[i].name != "" && P[i].wins >= 0 && P[i].points >= 0 && i < p_amount; i++) {
		writeStream << setw(2) << right << i + 1 << ") " << setw(20) << left << P[i].name << " " << setw(2) << right << P[i].wins << "  " << setw(5) << right << fixed << setprecision(2) << P[i].points << endl;
	}

	//I�vedama papildoma tu��ia eilut�
	writeStream << endl;

	return;
}

void WriteThreads(ThreadStruct P[], int p_amount, ofstream & writeStream) {

	//I�vedamos stulpeli� antra�t�s
	writeStream << " ----------------------------------\n";
	writeStream << setw(3) << "Nr." << " " << setw(60) << left << "Pavard�" << " " << setw(7) << "Pergal�s" << " " << setw(5) << "Ta�kai" << endl;

	//Kol yra tinkamai �vest� objekt� ir nepasiekia limito (array_limit), juos �ra�o � fail� pagal stulpelius
	for (int i = 0; P[i].name != "" && P[i].wins >= 0 && P[i].points >= 0 && i < p_amount; i++) {
		writeStream << setw(2) << right << i + 1 << ") " << setw(60+6) << left << P[i].name << " " << setw(2) << right << P[i].wins << "  " << setw(5) << right << fixed << setprecision(2) << P[i].points << endl;
	}

	//I�vedama papildoma tu��ia eilut�
	writeStream << endl;

	return;
}

void ExecuteThrust(ThreadStruct P1[], int p1_amount, ThreadStruct P2[], int p2_amount, ThreadStruct P3[], int p3_amount, ThreadStruct P4[], int p4_amount, ThreadStruct P5[], int p5_amount, ThreadStruct P[], int p_amount) {

	//Sukuriamas masyvas kiekvienam duomen� tipui atskirai
	string P_STRING[array_size];
	int P_INT[array_size] = { 0 };
	double P_DOUBLE[array_size] = { 0 };

	//Vykdomos sud�jimo funkcijos
	ExecuteStringAdd(P_STRING, P1, P2, P3, P4, P5);
	ExecuteIntAdd(P_INT, P1, P2, P3, P4, P5);
	ExecuteDoubleAdd(P_DOUBLE, P1, P2, P3, P4, P5);

	//Kopijuojamos reik�m�s i� tip� masyv� � bendr� strukt�r� P
	CopyTypeDataToArray(P_STRING, P_INT, P_DOUBLE, P);

}

void CopyArrayToTypeString(string P_STRING[], ThreadStruct P[]) {

	for (int i = 0; i < array_size; i++) {
		P_STRING[i] = P[i].name;
	}

}

void CopyArrayToTypeInt(int P_INT[], ThreadStruct P[]) {

	for (int i = 0; i < array_size; i++) {
		P_INT[i] = P[i].wins;
	}

}

void CopyArrayToTypeDouble(double P_DOUBLE[], ThreadStruct P[]) {

	for (int i = 0; i < array_size; i++) {
		P_DOUBLE[i] = P[i].points;
	}

}

void ExecuteStringAdd(string P_STRING[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]) {

	string P1_STRING[array_size];
	string P2_STRING[array_size];
	string P3_STRING[array_size];
	string P4_STRING[array_size];
	string P5_STRING[array_size];

	CopyArrayToTypeString(P1_STRING, P1);
	CopyArrayToTypeString(P2_STRING, P2);
	CopyArrayToTypeString(P3_STRING, P3);
	CopyArrayToTypeString(P4_STRING, P4);
	CopyArrayToTypeString(P5_STRING, P5);

	thrust::plus<string> operationToExecute;
	thrust::transform(P_STRING, P_STRING + array_size, P1_STRING, P_STRING, operationToExecute);
	thrust::transform(P_STRING, P_STRING + array_size, P2_STRING, P_STRING, operationToExecute);
	thrust::transform(P_STRING, P_STRING + array_size, P3_STRING, P_STRING, operationToExecute);
	thrust::transform(P_STRING, P_STRING + array_size, P4_STRING, P_STRING, operationToExecute);
	thrust::transform(P_STRING, P_STRING + array_size, P5_STRING, P_STRING, operationToExecute);

	//for (int i = 0; i < array_size; i++) {
	//	cout << P_STRING[i] << endl;
	//}

}

void ExecuteIntAdd(int P_INT[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]) {

	int P1_INT[array_size] = { 0 };
	int P2_INT[array_size] = { 0 };
	int P3_INT[array_size] = { 0 };
	int P4_INT[array_size] = { 0 };
	int P5_INT[array_size] = { 0 };

	CopyArrayToTypeInt(P1_INT, P1);
	CopyArrayToTypeInt(P2_INT, P2);
	CopyArrayToTypeInt(P3_INT, P3);
	CopyArrayToTypeInt(P4_INT, P4);
	CopyArrayToTypeInt(P5_INT, P5);

	thrust::plus<int> operationToExecute;
	thrust::transform(P_INT, P_INT + array_size, P1_INT, P_INT, operationToExecute);
	thrust::transform(P_INT, P_INT + array_size, P2_INT, P_INT, operationToExecute);
	thrust::transform(P_INT, P_INT + array_size, P3_INT, P_INT, operationToExecute);
	thrust::transform(P_INT, P_INT + array_size, P4_INT, P_INT, operationToExecute);
	thrust::transform(P_INT, P_INT + array_size, P5_INT, P_INT, operationToExecute);

}

void ExecuteDoubleAdd(double P_DOUBLE[], ThreadStruct P1[], ThreadStruct P2[], ThreadStruct P3[], ThreadStruct P4[], ThreadStruct P5[]) {

	double P1_DOUBLE[array_size] = { 0 };
	double P2_DOUBLE[array_size] = { 0 };
	double P3_DOUBLE[array_size] = { 0 };
	double P4_DOUBLE[array_size] = { 0 };
	double P5_DOUBLE[array_size] = { 0 };

	CopyArrayToTypeDouble(P1_DOUBLE, P1);
	CopyArrayToTypeDouble(P2_DOUBLE, P2);
	CopyArrayToTypeDouble(P3_DOUBLE, P3);
	CopyArrayToTypeDouble(P4_DOUBLE, P4);
	CopyArrayToTypeDouble(P5_DOUBLE, P5);

	thrust::plus<double> operationToExecute;
	thrust::transform(P_DOUBLE, P_DOUBLE + array_size, P1_DOUBLE, P_DOUBLE, operationToExecute);
	thrust::transform(P_DOUBLE, P_DOUBLE + array_size, P2_DOUBLE, P_DOUBLE, operationToExecute);
	thrust::transform(P_DOUBLE, P_DOUBLE + array_size, P3_DOUBLE, P_DOUBLE, operationToExecute);
	thrust::transform(P_DOUBLE, P_DOUBLE + array_size, P4_DOUBLE, P_DOUBLE, operationToExecute);
	thrust::transform(P_DOUBLE, P_DOUBLE + array_size, P5_DOUBLE, P_DOUBLE, operationToExecute);

}

void CopyTypeDataToArray(string P_STRING[], int P_INT[], double P_DOUBLE[], ThreadStruct P[]) {

	for (int i = 0; i < array_size; i++) {
		for(int j = 0; j < P_STRING[i].length(); j++)
			P[i].name[j] = P_STRING[i][j];
		P[i].wins = P_INT[i];
		P[i].points = P_DOUBLE[i];
	}

}

void CopyArrayToVector(thrust::host_vector<ThreadStruct> CPU_P, ThreadStruct P[]) {

	//Kopijuoja i� masyvo � vektori� visas reik�mes
	for (int i = 0; i < CPU_P.size(); i++) {
		//my_strcat(CPU_P[i].name, P[i].name);
		CPU_P[i].wins = P[i].wins;
		CPU_P[i].points = P[i].points;
	}

	//for (int i = 0; i < CPU_P.size(); i++) {
	//	cout << CPU_P[i].name << endl;
	//	cout << CPU_P[i].wins << endl;
	//	cout << CPU_P[i].points << endl;
	//}

}

void CopyVectorToArray(ThreadStruct P[], thrust::host_vector<ThreadStruct> CPU_P, int p_amount) {

	//Kopijuoja i� masyvo � vektori� visas reik�mes
	for (int i = 0; i < p_amount; i++) {
		//my_strcat(P[i].name, CPU_P[i].name);
		P[i].wins = CPU_P[i].wins;
		P[i].points = CPU_P[i].points;
	}

	//for (int i = 0; i < p_amount; i++) {
	//	cout << P[i].name << endl;
	//	cout << P[i].wins << endl;
	//	cout << P[i].points << endl;
	//}

}